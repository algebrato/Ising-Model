#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include"xorshift.h"
//#include"fill_ran.h"
using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define J 1
#define DIM 2
#define L 128
#define BLOCKL 16
#define GRIDL  (L/BLOCKL)
#define BLOCKS ((GRIDL*GRIDL)/2)
#define THREADS ((BLOCKL*BLOCKL)/2)
#define N (L*L)
#define TOT_TH  (BLOCKS*THREADS) 
#define END_SCALE       4294967296.0


typedef int spin_t;
typedef unsigned int UI;


__device__ float MTGPU(unsigned int *s1, unsigned int *s2, unsigned int *s3, unsigned int *s4){

	unsigned int x, y, z, w, t;

	x=*s1;
	y=*s2;
	z=*s3;
	w=*s4;
	
	for(int i=0; i<30; i++)
		t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	*s1 = x;
	*s2 = y;
	*s3 = z;
	*s4 = w;
	
	return w / END_SCALE;
}





texture<float,1,hipReadModeElementType> boltzT;

__global__ void do_update(spin_t *s_, UI *a, UI *b, UI *c, UI *d, UI offset){
	int tidx = threadIdx.x + blockDim.x*blockIdx.x;
	int tidmy= threadIdx.y + blockDim.y*blockIdx.y;
	int tidy = 2*tidmy+((tidx+offset)%2);
	
	int ide = s_[L*tidy+tidx]*(s_[L*tidy+((tidx==0)?L-1:tidx-1)]+s_[L*tidy+((tidx==L-1)?0:tidx+1)]+s_[L*((tidy==0)?L-1:tidy-1)+tidx]+s_[L*((tidy==L-1)?0:tidy+1)+tidx]);

	//Inizializzo i semi
	unsigned int *aa = &a[tidy+tidx];
	unsigned int *bb = &b[tidy+tidx];
	unsigned int *cc = &c[tidy+tidx];
	unsigned int *dd = &d[tidy+tidx];
	int ie=0;

	if(MTGPU(aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
		s_[L*tidy+tidx] = -s_[L*tidy+tidx];
		ie -=2*ide;
	}
	__syncthreads();
}


int main(int argc, char**argv){
	spin_t *s, *sD;
	UI *a, *a_d, *b, *b_d, *c, *c_d, *d, *d_d;

	dim3 grid(GRIDL, GRIDL/2);
	dim3 block(BLOCKL, BLOCKL/2);
	dim3 gridRES(GRIDL, GRIDL);
	dim3 blockRES(BLOCKL, BLOCKL);



	float BETA = atof(argv[3]);
	float boltzGPU[4*DIM+1];
	
	for(int i=-2*DIM; i<=2*DIM; i++){
		boltzGPU[i+2*DIM] = min(1.0,exp(-2*BETA*J*i));
	}

	float *boltzTex;
	hipMalloc((void**)&boltzTex, (4*DIM+1)*sizeof(float));
	hipMemcpy(boltzTex, boltzGPU,(4*DIM+1)*sizeof(float), H_D);
	hipBindTexture(NULL, boltzT, boltzTex, (4*DIM+1)*sizeof(float));

	a=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	b=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	c=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	d=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	

	//fill_ran_vec2(a, b, c, d, TOT_TH);


	hipMalloc((void**)&a_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&b_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&c_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&d_d, 2*TOT_TH*(sizeof(unsigned int)));

	hipMemcpy(a_d, a, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(b_d, b, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(c_d, c, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(d_d, d, TOT_TH*(sizeof(unsigned int)), H_D);
	

	s = (spin_t*)malloc(N*sizeof(spin_t));
	for(int i=0; i<N; i++) s[i]=1;
	hipMalloc((void**)&sD, N*sizeof(spin_t));
	hipMemcpy(sD, s, N*sizeof(spin_t), H_D);
	for(int i=0; i<10000; i++){
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 0);
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 1);
		hipDeviceSynchronize();

	}

	return 0;
	/* --> Tipica Domanda//Risposta
	   __device__ long d_answer;

	   __global__ void G_SearchByNameID() {
	     d_answer = 2;
	     }

	     int main() {
	     SearchByNameID<<<1,1>>>();
	     typeof(d_answer) answer;
	     hipMemcpyFromSymbol(&answer, HIP_SYMBOL("d_answer"), sizeof(answer), 0, hipMemcpyDeviceToHost);
	     printf("answer: %d\n", answer);
	     return 0;
	     }
		       */




}







