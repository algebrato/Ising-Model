#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include"xorshift.cu"
#include"fill_ran.h"
#include"get_time.h"

using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define J 1
#define DIM 2
#define L 128
#define BLOCKL 16
#define GRIDL  (L/BLOCKL)
#define BLOCKS ((GRIDL*GRIDL)/2)
#define THREADS ((BLOCKL*BLOCKL)/2)
#define N (L*L)
#define TOT_TH  (BLOCKS*THREADS) 


typedef int spin_t;
typedef unsigned int UI;


texture<float,1,hipReadModeElementType> boltzT;

__global__ void get_magnetization(spin_t *s_, float *vec_mag){
	__shared__ spin_t sS[(BLOCKL)*(BLOCKL+1)];
	__shared__ float sum_part;

	sS[threadIdx.x + BLOCKL*threadIdx.y] = s_[(blockIdx.x*BLOCKL + threadIdx.x )+(blockIdx.y*BLOCKL*L + threadIdx.y*L)];
	__syncthreads();
	if(threadIdx.x == 0)
		if(threadIdx.y == 0){
			sum_part=0;
			for(int i=0; i<BLOCKL; i++)
				for(int k=0; k<BLOCKL; k++)
					sum_part += sS[i*BLOCKL+k];
			__syncthreads();
			vec_mag[blockIdx.x+blockIdx.y*GRIDL] = sum_part / (BLOCKL*BLOCKL);
		}
}


__global__ void do_update(spin_t *s_, UI *a, UI *b, UI *c, UI *d, UI offset){
	int tidx = threadIdx.x + blockDim.x*blockIdx.x;
	int tidmy= threadIdx.y + blockDim.y*blockIdx.y;
	int tidy = 2*tidmy+((tidx+offset)%2);
	int ide = s_[L*tidy+tidx]*(s_[L*tidy+((tidx==0)?L-1:tidx-1)]+s_[L*tidy+((tidx==L-1)?0:tidx+1)]+s_[L*((tidy==0)?L-1:tidy-1)+tidx]+s_[L*((tidy==L-1)?0:tidy+1)+tidx]);

	//Inizializzo i semi
	unsigned int *aa = &a[tidy+tidx];
	unsigned int *bb = &b[tidy+tidx];
	unsigned int *cc = &c[tidy+tidx];
	unsigned int *dd = &d[tidy+tidx];
	int ie=0;

	if(MTGPU(aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
		s_[L*tidy+tidx] = -s_[L*tidy+tidx];
		ie -=2*ide;
	}
	__syncthreads();
	
	a[tidy+tidx] = *aa;
	b[tidy+tidx] = *bb;
	c[tidy+tidx] = *cc;
	d[tidy+tidx] = *dd;

	__syncthreads();
}

void get_lattice(spin_t *s_){
	for(int y=0; y<L; ++y){
		for(int x=0; x<L; ++x)
			printf("%i\t",s_[y*L+x]);
		printf("\n");
	}

}




int main(int argc, char**argv){
	spin_t *s, *sD;
	UI *a, *a_d, *b, *b_d, *c, *c_d, *d, *d_d;
	float *vec_mag, *vec_mag_d;

	dim3 grid(GRIDL, GRIDL);
	dim3 block(BLOCKL, BLOCKL/2);
	dim3 gridRES(GRIDL, GRIDL);
	dim3 blockRES(BLOCKL, BLOCKL);



	float BETA    = atof(argv[1]);
	int   STEP_MC = atoi(argv[2]);
	float boltzGPU[4*DIM+1];
	
	for(int i=-2*DIM; i<=2*DIM; i++){
		boltzGPU[i+2*DIM] = min(1.0,exp(-2*BETA*J*i));
	}

	float *boltzTex;
	hipMalloc((void**)&boltzTex, (4*DIM+1)*sizeof(float));
	hipMemcpy(boltzTex, boltzGPU,(4*DIM+1)*sizeof(float), H_D);
	hipBindTexture(NULL, boltzT, boltzTex, (4*DIM+1)*sizeof(float));

	a=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	b=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	c=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	d=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	

	fill_ran_vec2(a, b, c, d, TOT_TH);


	hipMalloc((void**)&a_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&b_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&c_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&d_d, 2*TOT_TH*(sizeof(unsigned int)));

	hipMemcpy(a_d, a, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(b_d, b, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(c_d, c, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(d_d, d, TOT_TH*(sizeof(unsigned int)), H_D);
	

	s = (spin_t*)malloc(N*sizeof(spin_t));
	for(int i=0; i<N; i++) s[i]=1;
	hipMalloc((void**)&sD, N*sizeof(spin_t));
	hipMemcpy(sD, s, N*sizeof(spin_t), H_D);


	vec_mag = (float*)malloc( (GRIDL*GRIDL)*sizeof(float));
	hipMalloc((void**)&vec_mag_d, (GRIDL*GRIDL)*sizeof(float));
	double m=0;
	double M=0;

	double start = getTime();
	for(int i=0; i < 1000; ++i){
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 0);
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 1);
	}



	for(int i=0; i < STEP_MC; ++i){
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 0);
		do_update<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 1);
		hipDeviceSynchronize();
		
		get_magnetization<<<gridRES, blockRES>>>(sD, vec_mag_d);
		hipMemcpy(vec_mag, vec_mag_d, (GRIDL*GRIDL)*sizeof(float), D_H);
		
		for(int bl=0; bl < (GRIDL*GRIDL); bl++ )
			m+=vec_mag[bl];
		m = m / (GRIDL*GRIDL);
		M+=m;
		m=0;
	}

	double end = getTime();

	M/=((double)STEP_MC);

	printf("%f\t%f\n", BETA, M);
	//printf("%i\t%f\n", L, (end-start)/((double)(L*L)*(STEP_MC)));
	
	return 0;

}







