#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include"xorshift.cu"
#include"fill_ran.h"
#include"get_time.h"
#include<time.h>
using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define J 1
#define DIM 2
#define L 4096
#define BLOCKL 16
#define GRIDL  (L/BLOCKL)
#define BLOCKS ((GRIDL*GRIDL)/2)
#define THREADS ((BLOCKL*BLOCKL)/2)
#define N (L*L)
#define TOT_TH  (BLOCKS*THREADS) 
#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]

typedef int spin_t;
typedef unsigned int UI;


texture<float,1,hipReadModeElementType> boltzT;

__global__ void get_magnetization(spin_t *s_, float *vec_mag){
	__shared__ spin_t sS[(BLOCKL)*(BLOCKL+1)];
	__shared__ float sum_part;

	sS[threadIdx.x + BLOCKL*threadIdx.y] = s_[(blockIdx.x*BLOCKL + threadIdx.x )+(blockIdx.y*BLOCKL*L + threadIdx.y*L)];
	__syncthreads();
	if(threadIdx.x == 0)
		if(threadIdx.y == 0){
			sum_part=0;
			for(int i=0; i<BLOCKL; i++)
				for(int k=0; k<BLOCKL; k++)
					sum_part += sS[i*BLOCKL+k];
			__syncthreads();
			vec_mag[blockIdx.x+blockIdx.y*GRIDL] = sum_part / (BLOCKL*BLOCKL);
		}
}


__global__ void do_update(spin_t *s_, UI *a, UI *b, UI *c, UI *d, UI offset, int *energie){
	int tidx = threadIdx.x + blockDim.x*blockIdx.x;
	int tidmy= threadIdx.y + blockDim.y*blockIdx.y;
	int tidy = 2*tidmy+(tidx+offset)%2;
	printf("%i\n",tidy);


	int ide = s_[L*tidy+tidx]*(s_[L*tidy+((tidx==0)?L-1:tidx-1)]+s_[L*tidy+((tidx==L-1)?0:tidx+1)]+s_[L*((tidy==0)?L-1:tidy-1)+tidx]+s_[L*((tidy==L-1)?0:tidy+1)+tidx]);

	//Inizializzo i semi
	unsigned int n = threadIdx.y*BLOCKL+threadIdx.x;
	//hiprandState localState = globalState[n];
	//float stopId = hiprand_uniform(&localState);
	unsigned int *aa = &a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *bb = &b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *cc = &c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *dd = &d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	
	__syncthreads();



	int ie=0;
	if(MTGPU(aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
		s_[L*tidy+tidx] = -s_[L*tidy+tidx];
		ie -=2*ide;
	}
	
	__syncthreads();
	
	//globalState[n] = localState;
	a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *aa;
	b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *bb;
	c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *cc;
	d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *dd;
	

	__shared__ int deltaE[THREADS];
	deltaE[n] = ie;
	for(int stride = THREADS>>1; stride > 0 ; stride >>=1){
		__syncthreads();
		if(n < stride) deltaE[n] += deltaE[n+stride];
	}
	if(n == 0) energie[blockIdx.y*GRIDL+blockIdx.x] += deltaE[0];		

	__syncthreads();
}

__global__ void do_update_testB(spin_t *s, UI *a, UI offset, int *energie){
	//Qui ci vanno un po' di variabili per inizializzare MTGPU
	unsigned int n = threadIdx.y*BLOCKL+threadIdx.x;
	unsigned int *aa = &a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	//Fine*************

	LCG32(aa);

	a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *aa;

}

__global__ void do_update_shared(spin_t *s, UI *a, UI *b, UI *c, UI *d, UI offset, int *energie){


	unsigned int n = threadIdx.y*BLOCKL+threadIdx.x;
	unsigned int Xoffset = blockIdx.x*BLOCKL;
	unsigned int Yoffset = (2*blockIdx.y+(blockIdx.x+offset)%2)*BLOCKL;

	__shared__ spin_t sS[(BLOCKL+2)*(BLOCKL+2)];

	//se non sono sui bordi completo con la doppia scacchiera
	sS[(2*threadIdx.y+1)*(BLOCKL+2)+threadIdx.x+1] = s[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)];
	sS[(2*threadIdx.y+2)*(BLOCKL+2)+threadIdx.x+1] = s[(Yoffset+2*threadIdx.y+1)*L+(Xoffset+threadIdx.x)];
	
	//bordo in alto
	/*if(threadIdx.y == 0)
		sS[threadIdx.x+1] = (Yoffset == 0) ? s[(L-1)*L+Xoffset+threadIdx.x] : s[(Yoffset-1)*L+Xoffset+threadIdx.x];
	if(threadIdx.y == (BLOCKL/2)-1)
		sS[(BLOCKL+1)*(BLOCKL+2)+(threadIdx.x+1)] = (Yoffset == L-BLOCKL) ? s[Xoffset+threadIdx.x] : s[(Yoffset+BLOCKL)*L+Xoffset+threadIdx.x];
	
	
	if(threadIdx.x == 0){
		if(blockIdx.x == 0){
			sS[(2*threadIdx.y+1)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y)*L+(L-1)];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y+1)*L+(L-1)];
		}
		else{
			sS[(2*threadIdx.y+1)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y)*L+(Xoffset-1)];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y+1)*L+(Xoffset-1)];
		}
	}

	if(threadIdx.x == BLOCKL-1){
		if(blockIdx.x == GRIDL-1){
			sS[(2*threadIdx.y+1)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y)*L];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y+1)*L];
		}
		else{
			sS[(2*threadIdx.y+1)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y)*L+Xoffset+BLOCKL];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y+1)*L+Xoffset+BLOCKL];
		}
	}
	__syncthreads();*/

	//Qui ci vanno un po' di variabili per inizializzare MTGPU
	unsigned int *aa = &a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *bb = &b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *cc = &c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *dd = &d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	//Fine*************
	int ie=0;
	unsigned int x = threadIdx.x;
	unsigned int y1= 2*threadIdx.y+(threadIdx.x%2);
        unsigned int y2= 2*threadIdx.y+((threadIdx.x+1)%2);
		
		int ide = sS(x,y1)*(sS(x-1,y1)+sS(x+1,y1)+sS(x,y1+1)+sS(x,y1-1));
		if(MTGPU(aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
			sS(x,y1) = -sS(x,y1);
			ie -=2*ide;
		}
		__syncthreads();
		
		ide = sS(x,y2)*(sS(x-1,y2)+sS(x+1,y2)+sS(x,y2+1)+sS(x,y2-1));
		if(MTGPU(aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
			sS(x,y2) = -sS(x,y2);
			ie -= 2*ide;
		}
		__syncthreads();

		s[(Yoffset+2*threadIdx.y)*L+Xoffset+threadIdx.x] = sS[(2*threadIdx.y+1)*(BLOCKL+2)+threadIdx.x+1];
		s[(Yoffset+2*threadIdx.y+1)*L+Xoffset+threadIdx.x] = sS[(2*threadIdx.y+2)*(BLOCKL+2)+threadIdx.x+1];
		a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *aa;
		b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *bb;
		c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *cc;
		d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *dd;
	
		__shared__ int deltaE[THREADS];
		deltaE[n] = ie;
		for(int stride = THREADS>>1; stride > 0 ; stride >>=1){
			__syncthreads();
			if(n < stride) deltaE[n] += deltaE[n+stride];
		}
		if(n == 0) energie[blockIdx.y*GRIDL+blockIdx.x] += deltaE[0];
		__syncthreads();

}



void get_lattice(spin_t *s_){
	for(int y=0; y<L; ++y){
		for(int x=0; x<L; ++x)
			printf("%i\t",s_[y*L+x]);
		printf("\n");
	}

}

int cpu_energy(spin_t *s){
        int ie = 0;
        for(int x = 0; x < L; ++x)
                for(int y = 0; y < L; ++y)
                        ie += s[L*y+x]*(s[L*y+((x==0)?L-1:x-1)]+s[L*y+((x==L-1)?0:x+1)]+s[L*((y==0)?L-1:y-1)+x]+s[L*((y==L-1)?0:y+1)+x]);
        return ie/2;
}


/*__global__ void setup_kernel ( hiprandState * state, unsigned long seed ){
	int id = threadIdx.x  + blockIdx.x + blockDim.x;
	hiprand_init ( seed, id , id, &state[id] );
}*/



int main(int argc, char**argv){
	hipSetDevice(1);
	spin_t *s, *sD;
	UI *a, *a_d, *b, *b_d, *c, *c_d, *d, *d_d;
	float *vec_mag, *vec_mag_d;
	int *energie, *energie_d;	
	time_t t;
	time(&t);

	dim3 grid(GRIDL, GRIDL/2);
	dim3 block(BLOCKL, BLOCKL/2);
	dim3 gridRES(GRIDL, GRIDL);
	dim3 blockRES(BLOCKL, BLOCKL);
	
	/*hiprandState* devStates;
	hipMalloc ( &devStates, TOT_TH*sizeof( hiprandState ) );
	setup_kernel <<< grid, block >>> ( devStates, (unsigned long) t );*/


	float BETA    = atof(argv[1]);
	int   STEP_MC = atoi(argv[2]);
	float boltzGPU[4*DIM+1];
	
	for(int i=-2*DIM; i<=2*DIM; i++){
		boltzGPU[i+2*DIM] = min(1.0,exp(-2*BETA*J*i));
	}

	float *boltzTex;
	hipMalloc((void**)&boltzTex, (4*DIM+1)*sizeof(float));
	hipMemcpy(boltzTex, boltzGPU,(4*DIM+1)*sizeof(float), H_D);
	hipBindTexture(NULL, boltzT, boltzTex, (4*DIM+1)*sizeof(float));

	a=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	b=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	c=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	d=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	

	fill_ran_vec4(a, b, c, d, TOT_TH);
	
	hipMalloc((void**)&a_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&b_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&c_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&d_d, 2*TOT_TH*(sizeof(unsigned int)));

	hipMemcpy(a_d, a, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(b_d, b, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(c_d, c, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(d_d, d, TOT_TH*(sizeof(unsigned int)), H_D);
	

	s = (spin_t*)malloc(N*sizeof(spin_t));
	for(int i=0; i<N; i++) s[i]=1;
	hipMalloc((void**)&sD, N*sizeof(spin_t));
	hipMemcpy(sD, s, N*sizeof(spin_t), H_D);


	vec_mag = (float*)malloc( (GRIDL*GRIDL)*sizeof(float));
	hipMalloc((void**)&vec_mag_d, (GRIDL*GRIDL)*sizeof(float));
	energie = (int*)malloc(BLOCKS*sizeof(int));
	hipMalloc((void**)&energie_d, BLOCKS*sizeof(int));	
		for(int i=0; i<BLOCKS; i++)
			energie[i]=0;
	hipMemcpy(energie_d, energie, BLOCKS*(sizeof(int)), H_D);

	int ie = cpu_energy(s);
	int sumE = ie;
	double E=0;
	double E_2=0;
	double m=0;
	double M=0;
	double chi_sqr=0;
	double chi_sqr_2=0;
	double chi_sqr_m=0;	
	
	for(int i=0; i < 1000; ++i){
		//do_update_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 0, energie_d);
		//do_update_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 1, energie_d);
		do_update_testB<<<grid, block>>>(sD, a_d, 0, energie_d);
		do_update_testB<<<grid, block>>>(sD, a_d, 1, energie_d);
	}


	double start = getTime();
	for(int i=0; i < STEP_MC; ++i){
		//do_update_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 0, energie_d);
		//do_update_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, 1, energie_d);
		do_update_testB<<<grid, block>>>(sD, a_d, 0, energie_d);
		do_update_testB<<<grid, block>>>(sD, a_d, 1, energie_d);
			
		hipDeviceSynchronize();
		
		//get_magnetization<<<gridRES, blockRES>>>(sD, vec_mag_d);
		//hipMemcpy(vec_mag, vec_mag_d, (GRIDL*GRIDL)*sizeof(float), D_H);
		
		//for(int bl=0; bl < (GRIDL*GRIDL); bl++ )
		//	m+=vec_mag[bl];
		//m = m / (GRIDL*GRIDL);
		//M+=fabs(m);
		
		//hipMemcpy(energie, energie_d, BLOCKS*sizeof(int), D_H);
		//for(int bl=0; bl < BLOCKS; bl++)
		//	sumE+=energie[bl];
		//E += (double)sumE;
		//chi_sqr+=pow(E/(i+1)-sumE,2.);
		//E_2 += pow((double)sumE,2.);
		//chi_sqr_2+=pow(E_2/(i+1)-pow((double)sumE,2.),2.);
		//chi_sqr_m+=pow(M/(i+1)-fabs(m),2.);			
		
		//m=0;	
		//sumE=ie;
	}
	double end = getTime();
	//hipMemcpy(s, sD, N*sizeof(spin_t), D_H);
	//get_lattice(s);
	
	E_2 /= (long double)STEP_MC;
	E   /= (double)STEP_MC;
	double Cal_Spec=(1/((double)N))*(E_2-E*E)*((double)BETA*(double)BETA);
	double sigma_E  = pow(chi_sqr/((double)STEP_MC),0.5);
	double sigma_E2 = pow(chi_sqr_2/((double)STEP_MC),0.5);
	double sigma_m  = pow(chi_sqr_m/((double)STEP_MC),0.5);
	double err_per=0.5*(sigma_E/E+sigma_E2/E_2);
		

	M/=((double)STEP_MC);

	//printf("%f\t%f\t%f\t%f\t%f\n", BETA, M, Cal_Spec, err_per*Cal_Spec, sigma_m);
	printf("%i\t%f\n", L, (end-start)/((double)(L*L)*(STEP_MC)));
	
	return 0;

}







