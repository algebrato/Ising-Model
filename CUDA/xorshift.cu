
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#define END_SCALE       4294967296.0F


__device__ __host__ float MTGPU(unsigned int *s1, unsigned int *s2, unsigned int *s3, unsigned int *s4){

	unsigned int x, y, z, w, t;

	x=*s1;
	y=*s2;
	z=*s3;
	w=*s4;

	//for(int i=0; i<50; i++) //il mix potrebbe in questo caso non essere una buona idea.
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	*s1 = x;
	*s2 = y;
	*s3 = z;	
	*s4 = w;

	return w / END_SCALE;
}

	

