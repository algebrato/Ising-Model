#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#include<hip/hip_runtime_api.h>
#include<time.h>
#define END_SCALE 4294967296.0F


using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define L 516
#define BLOCKL 16


__global__ void get_max(float *arr_num, float ABSMAX){
	__shared__ float sub[BLOCKL];
	__shared__ float subsub[L/BLOCKL];
	sub[threadIdx.x] = arr_num[BLOCKL*blockIdx.x+threadIdx.x];
	__shared__ float max;
	ABSMAX=0;

	max = sub[0];
	for(int i=1; i<BLOCKL; ++i){
		if(max < sub[i]){
			max = sub[i];
		}
	}
	subsub[blockIdx.x]=max;
	__syncthreads();
	
	for(int k=0; k<L/BLOCKL; ++k){
		if(&ABSMAX < subsub[k]){
			&ABSMAX = subsub[k];
		}
	}
}





int main(){
	float *arr_num, *arr_num_d, *ABSMAX_d, *ABSMAX;
	
	//Dichiaro la mia array sulla CPU
	arr_num=(float*)malloc(L*sizeof(float));
	ABSMAX=(float*)malloc(sizeof(float));	
	//Riempio l'array con numeri random tra 0 e 10
	for(int i=0; i<L; ++i){
		arr_num[i]=10*(rand()/float(RAND_MAX));
		//printf("%f\n",arr_num[i]);
	}

	dim3 grid (L/BLOCKL);
	dim3 block(BLOCKL);


	//Copio l'array sulla scheda grafica
	hipMalloc((void**)&arr_num_d, L*sizeof(float));
	hipMalloc((void**)&ABSMAX_d, sizeof(float));
	hipMemcpy(arr_num_d, arr_num, L*sizeof(float), H_D);

	get_max<<< grid , block >>>(arr_num_d,ABSMAX_d);



	hipMemcpy(ABSMAX, ABSMAX_d, sizeof(float), D_H);

	cout << "Max = " << ABSMAX << endl;


	return 0;
}






