
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand/hiprand.h>
#include<hip/hip_runtime_api.h>
#include<time.h>
#define END_SCALE 4294967296.0F


using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define L 516
#define BLOCKL 16


__global__ void get_max(float *arr_num, float *sub_arr){
	__shared__ float sub[BLOCKL];
	sub[threadIdx.x] = arr_num[BLOCKL*blockIdx.x+threadIdx.x];
	__shared__ float max;
	max = sub[0];
	for(int i=1; i<BLOCKL; ++i){
		if(max < sub[i]){
			max = sub[i];
		}
	}
	sub_arr[blockIdx.x]=max;
}





int main(){
	float *arr_num, *arr_num_d, *sub_arr_d, *sub_arr;
	
	//Dichiaro la mia array sulla CPU
	sub_arr=(float*)malloc((L/BLOCKL)*sizeof(float));
	arr_num=(float*)malloc(L*sizeof(float));
	
	//Riempio l'array con numeri random tra 0 e 10
	for(int i=0; i<L; ++i){
		arr_num[i]=10*(rand()/float(RAND_MAX));
		//printf("%f\n",arr_num[i]);
	}

	dim3 grid (L/BLOCKL);
	dim3 block(BLOCKL);


	//Copio l'array sulla scheda grafica
	hipMalloc((void**)&arr_num_d, L*sizeof(float));
	hipMalloc((void**)&sub_arr_d, (L/BLOCKL)*sizeof(float));
	hipMemcpy(arr_num_d, arr_num, L*sizeof(float), H_D);

	get_max<<< grid , block >>>(arr_num_d,sub_arr_d);



	hipMemcpy(sub_arr, sub_arr_d, (L/BLOCKL)*sizeof(float), D_H);

	cout << "Result" << endl;
	for(int i = 0 ; i<(L/BLOCKL);i++)
		cout << sub_arr[i] << endl;


	return 0;
}






