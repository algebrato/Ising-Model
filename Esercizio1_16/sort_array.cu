#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>
#include<hip/hip_runtime_api.h>
#include<time.h>
#define END_SCALE 4294967296.0F


using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define L 4096
#define BLOCKL 16


__global__ void get_max(float *arr_num, float *sub_arr){
	__shared__ float sub[BLOCKL];
	sub[threadIdx.x] = arr_num[BLOCKL*blockIdx.x+threadIdx.x];
	__shared__ float max;

	max = sub[0];
	for(int i=1; i<BLOCKL; ++i){
		if(max < sub[i]){
			max = sub[i];
		}
	}
	sub_arr[blockIdx.x]=max;
	__syncthreads();
}


int main(){
	float *arr_num, *arr_num_d, *sub_arr_d, *sub_arr;
	
	//Dichiaro la mia array sulla CPU
	arr_num=(float*)malloc(L*sizeof(float));
	sub_arr=(float*)malloc((L/BLOCKL)*sizeof(float));	
	//Riempio l'array con numeri random tra 0 e 10
	for(int i=0; i<L; ++i){
		arr_num[i]=10*(rand()/float(RAND_MAX));
		//printf("%f\n",arr_num[i]);
	}

	dim3 grid (L/BLOCKL);
	dim3 block(BLOCKL);


	//Copio l'array sulla scheda grafica
	hipMalloc((void**)&arr_num_d, L*sizeof(float));
	hipMalloc((void**)&sub_arr_d, (L/BLOCKL)*sizeof(float));
	hipMemcpy(arr_num_d, arr_num, L*sizeof(float), H_D);

	get_max<<< L/BLOCKL , block >>>(arr_num_d,sub_arr_d);
	get_max<<< L/(BLOCKL*16) , block >>>(sub_arr_d,arr_num_d);
	get_max<<< L/(BLOCKL*256) , block >>>(arr_num_d,sub_arr_d);
	
	
	hipMemcpy(sub_arr, sub_arr_d, (L/BLOCKL)*sizeof(float), D_H);

	cout << "Massimo array = " << sub_arr[i] << endl;

	return 0;
}

