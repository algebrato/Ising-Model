/*L'idea base del programma e` quella di spezzare il vettore principale
  in tanti piccoli vettori di dimensione 16. Trovare il massimo di ognuno
  di questi e scriverlo in un sottovettore che e` sub_array.
  Ogni volta che viene ripetuto il kernel la lunghezza di sub_array, rispetto
  ad array e` 16 volte inferiore.
  Se si parte da un vettore iniziale di 4096 elementi non e` necessario nemmeno
  usare la CPU. Iterando 3 volte il kernel, torna una matrice di dimensione 1 
  che rappresenta appunto il massimo del vettore iniziale
 */

#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand/hiprand.h>
#include<hip/hip_runtime_api.h>
#include<time.h>
#define END_SCALE 4294967296.0F


using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define L 4096
#define BLOCKL 16


__global__ void get_max(float *arr_num, float *sub_arr){
	__shared__ float sub[BLOCKL];
	sub[threadIdx.x] = arr_num[BLOCKL*blockIdx.x+threadIdx.x];
	__shared__ float max;

	max = sub[0];
	for(int i=1; i<BLOCKL; ++i){
		if(max < sub[i]){
			max = sub[i];
		}
	}
	sub_arr[blockIdx.x]=max;
	__syncthreads();
}


int main(){
		

	float *arr_num, *arr_num_d, *sub_arr_d, *sub_arr;
	
	//Alloco la memoria per le array sulla CPU
	arr_num=(float*)malloc(L*sizeof(float));
	sub_arr=(float*)malloc((L/BLOCKL)*sizeof(float));	

	//Riempio l'array con numeri random tra 0 e 10
	for(int i=0; i<L; ++i){
		arr_num[i]=10*(rand()/float(RAND_MAX));
	}
	
	//Alloco la memoria per le array sulla GPU
	hipMalloc((void**)&arr_num_d, L*sizeof(float));
	hipMalloc((void**)&sub_arr_d, (L/BLOCKL)*sizeof(float));
	
	//Copio l'array di numeri random sulla GPU
	hipMemcpy(arr_num_d, arr_num, L*sizeof(float), H_D);

	//Eseguo 3 volte il kernel invertendo arr_num e sub_arr in questo modo
	//ad ogni step la dimensione dell'array dei massimi dei sottovettori
	//allocati nella memoria shared, diminuisce di un fattore 16
	//4096 e` una potenza perfetta di 16, quindi dopo 3 passaggi
	//ritorneta` una matrice dei massimi di dimensione 1, che rappresentera`
	//proprio il massimo del vettore iniziale.
	get_max<<< L/BLOCKL , BLOCKL >>>(arr_num_d,sub_arr_d);
	get_max<<< L/(BLOCKL*16) , BLOCKL >>>(sub_arr_d,arr_num_d);
	get_max<<< L/(BLOCKL*256) , BLOCKL >>>(arr_num_d,sub_arr_d);
	
		
	hipMemcpy(sub_arr, sub_arr_d, (L/BLOCKL)*sizeof(float), D_H);

	cout << "Massimo array = " << sub_arr[0] << endl;

	return 0;
}

