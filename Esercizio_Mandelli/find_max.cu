/*L'idea base del programma e` quella di spezzare il vettore principale
  in tanti piccoli vettori di dimensione 16. Trovare il massimo di ognuno
  di questi e scriverlo in un sottovettore che e` sub_array.
  Ogni volta che viene ripetuto il kernel la lunghezza di sub_array, rispetto
  ad array e` 16*BLOCKL volte inferiore.
  Se si parte da un vettore iniziale di 4096 elementi non e` necessario nemmeno
  usare la CPU. Iterando 3 volte il kernel, torna un vettore di dimensione 1 
  che rappresenta appunto il massimo del vettore iniziale
 */

#include <hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand/hiprand.h>
#include<hip/hip_runtime_api.h>
#include<time.h>
#define END_SCALE 4294967296.0F


using namespace std;


#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost

#define L 4096
#define BLOCKL 16


__global__ void get_max(float *arr_num, float *sub_arr){
	//Alloco nella memoria shared uno spazio dedicato 
	//ai sottovettori di dimensione 16 (dimensione del warp di una GTX480)
	__shared__ float sub[BLOCKL];

	//Riempio i vettori nella memoria shared
	sub[threadIdx.x] = arr_num[BLOCKL*blockIdx.x+threadIdx.x];
	
	//ad ogni blocco di memoria shared e` associato il relativo massimo
	__shared__ float max;

	//calcolo il massimo (in modo sequenziale) per ogni blocco
	max = sub[0];
	for(int i=1; i<BLOCKL; ++i){
		if(max < sub[i]){
			max = sub[i];
		}
	}
	//riempio il vettore dei MASSIMI. Ogni elemento di questo vettore contiene il massimo dei 16
	//elementi del vettore stanziato inizialmente. Rispetto al vettore iniziale
	//sara` piu` corto di un fattore 16*BLOCKL
	sub_arr[blockIdx.x]=max;

	//aspetto che tutti i threads dei vari blocchi abbiano finito
	__syncthreads();
}


int main(){
		

	float *arr_num, *arr_num_d, *sub_arr_d, *sub_arr;
	
	//Alloco la memoria per le array sulla CPU
	arr_num=(float*)malloc(L*sizeof(float));
	sub_arr=(float*)malloc((L/BLOCKL)*sizeof(float));	

	//Riempio l'array con numeri random tra 0 e 10
	for(int i=0; i<L; ++i){
		arr_num[i]=10*(rand()/float(RAND_MAX));
	}
	
	//Alloco la memoria per le array sulla GPU
	hipMalloc((void**)&arr_num_d, L*sizeof(float));
	hipMalloc((void**)&sub_arr_d, (L/BLOCKL)*sizeof(float));
	
	//Copio l'array di numeri random sulla GPU
	hipMemcpy(arr_num_d, arr_num, L*sizeof(float), H_D);

	//Eseguo 3 volte il kernel invertendo arr_num e sub_arr in questo modo
	//ad ogni step la dimensione dell'array dei massimi dei sottovettori
	//allocati nella memoria shared, diminuisce di un fattore 16*BLOCKL.
	//4096 e` una potenza di 16, quindi dopo 3 passaggi
	//ritorneta` un vettore dei massimi di dimensione 1, che rappresentera`
	//proprio il massimo del vettore iniziale.
	get_max<<< L/BLOCKL , BLOCKL >>>(arr_num_d,sub_arr_d);
	get_max<<< L/(BLOCKL*16) , BLOCKL >>>(sub_arr_d,arr_num_d);
	get_max<<< L/(BLOCKL*256) , BLOCKL >>>(arr_num_d,sub_arr_d);
	
		
	hipMemcpy(sub_arr, sub_arr_d, (L/BLOCKL)*sizeof(float), D_H);

	cout << "Massimo array = " << sub_arr[0] << endl;

	return 0;
}

