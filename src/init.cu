
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>


void Info_Dev(int i){

        hipDeviceProp_t prop;
        int count;

        hipGetDeviceCount (&count);

        hipGetDeviceProperties( &prop, i );
	
	printf("Outfit del device in uso:\n");
        printf("\n");
        printf("Device Name:\t\t %s\n",prop.name);
        printf("Shared Mem/block :\t %d\n",prop.sharedMemPerBlock);
        printf("Registri per blocco:\t %d\n",prop.regsPerBlock);
        printf("Warp size:\t\t %d\n",prop.warpSize);
        printf("Texature 1D :\t\t %d\n",prop.maxTexture1D);
        printf("\n");

}
