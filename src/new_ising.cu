#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include "render.h"
#include "init.h"
#include "fill_ran.h"

#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost
#define DEV 0

#define DIM     2                       //Primi vicini mezzi
#define L       1024                    //Dimensione del reticolo
#define BLOCKL  16                      //Threads per blocco
#define GRIDL   (L/BLOCKL)              //N. Blocchi lineari (per colonna?) 
#define BLOCKS  ((GRIDL*GRIDL)/2)       //N. Blocchi totali nel reticolo
#define THREADS ((BLOCKL*BLOCKL)/2)     //N. Threads per blocco
#define N       (L*L)                   //N. Celle del reticolo
#define SPINS_PER_BLOCK (N/2)           //NON HA SENSO! (togliere questa follia)
#define TOT_TH  (BLOCKS*THREADS)        //N. di threads totali della griglia.

#define STEP            1000
#define TERM_STEP       100
#define VUOTO           1000
#define TOT             (STEP*VUOTO)
#define J 1
#define END_SCALE	4294967296.0F

#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]
#define A  1664525
#define C  1013904223
#define AA A
#define CC C
#define MULT 2.328306437080797e-10f
#define MULT2 4.6566128752457969e-10f
#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]
#define RAN(n) (n = AA*n + CC)



typedef int spin_t;
typedef unsigned int UI;

texture<float,1,hipReadModeElementType> boltzT;

__device__ float MTGPU(unsigned int tid, unsigned int *s1, unsigned int *s2, unsigned int *s3, unsigned int *s4){
	unsigned int x, y, z, w, t;
	
	x=*s1;
	y=*s2;
	z=*s3;
	w=*s4;

	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;

	*s1 = x;
	*s2 = y;
	*s3 = z;
	*s4 = w;

	return w / END_SCALE;

}

__global__ void Comp_Res(spin_t *s, float *result){

        __shared__ spin_t sS[(BLOCKL)*(BLOCKL+1)];
        __shared__ float sum_part;

        sS[threadIdx.x + BLOCKL*threadIdx.y] = s[(blockIdx.x*BLOCKL + threadIdx.x )+(blockIdx.y*BLOCKL*L + threadIdx.y*L)];
        __syncthreads();


        if(threadIdx.x == 0)
                if(threadIdx.y == 0){
                        sum_part=0;
                        for(int i=0; i<BLOCKL; i++)
                                for(int k=0; k<BLOCKL; k++)
                                        sum_part += sS[i*BLOCKL+k];
                        __syncthreads();
                        result[blockIdx.x+blockIdx.y*GRIDL] = sum_part / (BLOCKL*BLOCKL);
                }
}








__global__ void update_metropolis_shared(spin_t *s, unsigned int* a, unsigned int* b, unsigned int* c, unsigned int* d, int* energie, unsigned int offset){
//__global__ void update_metropolis_shared(spin_t *s, int *ranvec, int* energie, unsigned int offset, hiprandStateMRG32k3a *state){


	unsigned int n = threadIdx.y*BLOCKL+threadIdx.x;
	unsigned int Xoffset = blockIdx.x*BLOCKL;
	unsigned int Yoffset = (2*blockIdx.y+(blockIdx.x+offset)%2)*BLOCKL;

	__shared__ spin_t sS[(BLOCKL+2)*(BLOCKL+2)];

	//se non sono sui bordi completo con la doppia scacchiera
	sS[(2*threadIdx.y+1)*(BLOCKL+2)+threadIdx.x+1] = s[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)];
	sS[(2*threadIdx.y+2)*(BLOCKL+2)+threadIdx.x+1] = s[(Yoffset+2*threadIdx.y+1)*L+(Xoffset+threadIdx.x)];
	
	//bordo in alto
	if(threadIdx.y == 0)
		sS[threadIdx.x+1] = (Yoffset == 0) ? s[(L-1)*L+Xoffset+threadIdx.x] : s[(Yoffset-1)*L+Xoffset+threadIdx.x];
	if(threadIdx.y == (BLOCKL/2)-1)
		sS[(BLOCKL+1)*(BLOCKL+2)+(threadIdx.x+1)] = (Yoffset == L-BLOCKL) ? s[Xoffset+threadIdx.x] : s[(Yoffset+BLOCKL)*L+Xoffset+threadIdx.x];
	
	
	if(threadIdx.x == 0){
		if(blockIdx.x == 0){
			sS[(2*threadIdx.y+1)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y)*L+(L-1)];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y+1)*L+(L-1)];
		}
		else{
			sS[(2*threadIdx.y+1)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y)*L+(Xoffset-1)];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)] = s[(Yoffset+2*threadIdx.y+1)*L+(Xoffset-1)];
		}
	}

	if(threadIdx.x == BLOCKL-1){
		if(blockIdx.x == GRIDL-1){
			sS[(2*threadIdx.y+1)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y)*L];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y+1)*L];
		}
		else{
			sS[(2*threadIdx.y+1)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y)*L+Xoffset+BLOCKL];
			sS[(2*threadIdx.y+2)*(BLOCKL+2)+BLOCKL+1] = s[(Yoffset+2*threadIdx.y+1)*L+Xoffset+BLOCKL];
		}
	}
	__syncthreads();

	//Qui ci vanno un po' di variabili per inizializzare MTGPU
	unsigned int *aa = &a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *bb = &b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *cc = &c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *dd = &d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	//Fine*************
	int ie=0;
	unsigned int x = threadIdx.x;
	unsigned int y1= 2*threadIdx.y+(threadIdx.x%2);
        unsigned int y2= 2*threadIdx.y+((threadIdx.x+1)%2);
		
		int ide = sS(x,y1)*(sS(x-1,y1)+sS(x+1,y1)+sS(x,y1+1)+sS(x,y1-1));
		if(MTGPU(n, aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
			sS(x,y1) = -sS(x,y1);
			ie -=2*ide;
		}
		__syncthreads();
		
		ide = sS(x,y2)*(sS(x-1,y2)+sS(x+1,y2)+sS(x,y2+1)+sS(x,y2-1));
		if(MTGPU(n, aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
			sS(x,y2) = -sS(x,y2);
			ie -= 2*ide;
		}
		__syncthreads();

		s[(Yoffset+2*threadIdx.y)*L+Xoffset+threadIdx.x] = sS[(2*threadIdx.y+1)*(BLOCKL+2)+threadIdx.x+1];
		s[(Yoffset+2*threadIdx.y+1)*L+Xoffset+threadIdx.x] = sS[(2*threadIdx.y+2)*(BLOCKL+2)+threadIdx.x+1];
		a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *aa;
		b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *bb;
		c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *cc;
		d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *dd;
	
		__shared__ int deltaE[THREADS];
		deltaE[n] = ie;
		for(int stride = THREADS>>1; stride > 0 ; stride >>=1){
			__syncthreads();
			if(n < stride) deltaE[n] += deltaE[n+stride];
		}
		if(n == 0) energie[blockIdx.y*GRIDL+blockIdx.x] += deltaE[0];
		__syncthreads();

}


int cpu_energy(spin_t *s){
	int ie = 0;
	for(int x = 0; x < L; ++x)
		for(int y = 0; y < L; ++y)
			ie += s[L*y+x]*(s[L*y+((x==0)?L-1:x-1)]+s[L*y+((x==L-1)?0:x+1)]+s[L*((y==0)?L-1:y-1)+x]+s[L*((y==L-1)?0:y+1)+x]);
	return ie/2;
}

int main(int pippo, char **pero){

	spin_t *s, *sD;

	float *risultati, *risultatiD;
	int *energie, *energieD;

	//Magnetizzazzione (ora non usata)
	risultati = (float*)malloc( (GRIDL*GRIDL)*sizeof(float));
	hipMalloc((void**)&risultatiD, (GRIDL*GRIDL)*sizeof(float));

	//Valori delle energie
	energie = (int*)malloc(BLOCKS*sizeof(int));
	hipMalloc((void**)&energieD, BLOCKS*sizeof(int));
	for(int i=0; i<BLOCKS; i++)
		energie[i]=0;
	hipMemcpy(energieD, energie, BLOCKS*(sizeof(int)), H_D);

	//Vettore di numeri inizialmente random
	unsigned int *a, *a_d, *b, *b_d, *c, *c_d, *d, *d_d;
	float BETA = atof (pero[3]);
	
	float boltzGPU[4*DIM+1];
	for(int i=-2*DIM; i<=2*DIM; i++) boltzGPU[i+2*DIM] = min(1.0,exp(-2*BETA*J*i));
        float *boltzTex;
	hipMalloc((void**)&boltzTex, (4*DIM+1)*sizeof(float));
	hipMemcpy(boltzTex, boltzGPU,(4*DIM+1)*sizeof(float), H_D);
	hipBindTexture(NULL, boltzT, boltzTex, (4*DIM+1)*sizeof(float));

	dim3 grid(GRIDL, GRIDL/2);
	dim3 block(BLOCKL, BLOCKL/2);

	dim3 gridRES(GRIDL, GRIDL);
	dim3 blockRES(BLOCKL, BLOCKL);

	//*************Accocazione e Generazione seed iniziali                 
	a=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	b=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	c=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	d=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	
	fill_ran_vec(a, b, c, d, TOT_TH);

	hipMalloc((void**)&a_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&b_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&c_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&d_d, 2*TOT_TH*(sizeof(unsigned int)));

	hipMemcpy(a_d, a, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(b_d, b, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(c_d, c, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(d_d, d, TOT_TH*(sizeof(unsigned int)), H_D);

	//*********Fine parte di generazione seed


	s = (spin_t*)malloc(N*sizeof(spin_t));
	for(int i=0; i<N; i++) s[i]=1;

	hipMalloc((void**)&sD, N*sizeof(spin_t));
	hipMemcpy(sD, s, N*sizeof(spin_t), H_D);
	int ie = cpu_energy(s); // Energia Iniziale.

	//printf("Energia iniziale: %i\n",ie);


	for(int i=0; i<TERM_STEP; i++){
		update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 0);
		update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 1);
		hipDeviceSynchronize();
	}


	double M=0;
	double Magnetizzazione=0;
	double E = ie;	
	int sumE=0;
	double E_2=0;
	int sub_iter=0;
	for(int i=0; i<STEP; i++){
		for(int j=0; j<VUOTO; j++){
	                update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 0);
	                update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 1);
			hipDeviceSynchronize();
		}
		sub_iter+=VUOTO;
		Comp_Res<<<gridRES, blockRES>>>(sD, risultatiD);
		hipMemcpy(risultati, risultatiD, (GRIDL*GRIDL)*sizeof(float), D_H);

		for(int bl=0; bl < (GRIDL*GRIDL); bl++ )
			M+=risultati[bl];
		M = M / (GRIDL*GRIDL);
		
		//printf("%f\n",M);

		hipMemcpy(energie, energieD, BLOCKS*sizeof(int), D_H);
		for(int bl=0; bl < BLOCKS; bl++)
			E+=energie[bl];
		sumE += E;
		E_2 += pow(E,2.);
		E=ie;
		
		Magnetizzazione +=M;
		M=0;
	}


	E_2 = E_2/STEP;
	E = sumE/STEP ;
	Magnetizzazione/=STEP;
	
	
	
	
	
	
	
	
	
	
	if(Magnetizzazione < 0)
		Magnetizzazione*=-1;

	printf("%f\t\t%f\t\t%f\t%f\n", BETA, Magnetizzazione, E, E_2);




	hipFree(sD);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(energieD);



	return 0;



}
