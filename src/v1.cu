#include "hip/hip_runtime.h"
#include<stdlib.h>
#include<stdio.h>
#include "render.h"
#include "init.h"
#include "fill_ran.h"

#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#define H_D hipMemcpyHostToDevice
#define D_H hipMemcpyDeviceToHost
#define DEV 0

#define DIM     2                       //Primi vicini mezzi
#define L       1024                    //Dimensione del reticolo
#define BLOCKL  16                      //Threads per blocco
#define GRIDL   (L/BLOCKL)              //N. Blocchi lineari (per colonna?) 
#define BLOCKS  ((GRIDL*GRIDL)/2)       //N. Blocchi totali nel reticolo
#define THREADS ((BLOCKL*BLOCKL)/2)     //N. Threads per blocco
#define N       (L*L)                   //N. Celle del reticolo
#define SPINS_PER_BLOCK (N/2)           //NON HA SENSO! (togliere questa follia)
#define TOT_TH  (BLOCKS*THREADS)        //N. di threads totali della griglia.

#define STEP            10
#define TERM_STEP       100
#define VUOTO           1000
#define TOT             (STEP*VUOTO)
#define J 1
#define END_SCALE	4294967296.0F

#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]
#define A  1664525
#define C  1013904223
#define AA A
#define CC C
#define MULT 2.328306437080797e-10f
#define MULT2 4.6566128752457969e-10f
#define sS(x,y) sS[(y+1)*(BLOCKL+2)+x+1]
#define RAN(n) (n = AA*n + CC)



typedef int spin_t;
typedef unsigned int UI;

texture<float,1,hipReadModeElementType> boltzT;

__device__ float MTGPU(unsigned int tid, unsigned int *s1, unsigned int *s2, unsigned int *s3, unsigned int *s4){
	unsigned int x, y, z, w, t;
	
	x=*s1;
	y=*s2;
	z=*s3;
	w=*s4;

	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
    	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;
	t=x^x<<11;x=y;y=z;z=w;w^=w>>19^t^t>>8;

 
	*s1 = x;
	*s2 = y;
	*s3 = z;
	*s4 = w;

	return w / END_SCALE;

}

__global__ void Comp_Res(spin_t *s, float *result){

        __shared__ spin_t sS[(BLOCKL)*(BLOCKL+1)];
        __shared__ float sum_part;

        sS[threadIdx.x + BLOCKL*threadIdx.y] = s[(blockIdx.x*BLOCKL + threadIdx.x )+(blockIdx.y*BLOCKL*L + threadIdx.y*L)];
        __syncthreads();


        if(threadIdx.x == 0)
                if(threadIdx.y == 0){
                        sum_part=0;
                        for(int i=0; i<BLOCKL; i++)
                                for(int k=0; k<BLOCKL; k++)
                                        sum_part += sS[i*BLOCKL+k];
                        __syncthreads();
                        result[blockIdx.x+blockIdx.y*GRIDL] = sum_part / (BLOCKL*BLOCKL);
                }
}








__global__ void update_metropolis_shared(spin_t *si, unsigned int* a, unsigned int* b, unsigned int* c, unsigned int* d, int* energie, unsigned int offset){

	unsigned int n = threadIdx.y*BLOCKL+threadIdx.x;
	unsigned int Xoffset = blockIdx.x*BLOCKL;
	unsigned int Yoffset = (2*blockIdx.y+(blockIdx.x+offset)%2)*BLOCKL;
	

	__global__ spint_t sf[L];

	int ie=0;
	unsigned int *aa = &a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *bb = &b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *cc = &c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];
	unsigned int *dd = &d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n];


	int ide = si[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)]*(si[(Yoffset+2*threadIdx.y)*(L+1)+(Xoffset+threadIdx.x)]+si[(Yoffset+2*threadIdx.y)*(L-1)+(Xoffset+threadIdx.x)]+si[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x+1)]+si[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x-1)]);
	if(MTGPU(n, aa, bb, cc, dd) < tex1Dfetch(boltzT, ide+2*DIM)){
		sf[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)] = -si[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)];
		ie -=2*ide;
	}
	__syncthreads();

	a[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *aa;
	b[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *bb;
	c[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *cc;
	d[(blockIdx.y*GRIDL+blockIdx.x)*THREADS+n] = *dd;
	si[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)] = sf[(Yoffset+2*threadIdx.y)*L+(Xoffset+threadIdx.x)];
	
	__syncthreads();

}


int cpu_energy(spin_t *s){
	int ie = 0;
	for(int x = 0; x < L; ++x)
		for(int y = 0; y < L; ++y)
			ie += s[L*y+x]*(s[L*y+((x==0)?L-1:x-1)]+s[L*y+((x==L-1)?0:x+1)]+s[L*((y==0)?L-1:y-1)+x]+s[L*((y==L-1)?0:y+1)+x]);
	return ie/2;
}

int main(int pippo, char **pero){

	spin_t *s, *sD;

	float *risultati, *risultatiD;
	int *energie, *energieD;

	//Magnetizzazzione (ora non usata)
	risultati = (float*)malloc( (GRIDL*GRIDL)*sizeof(float));
	hipMalloc((void**)&risultatiD, (GRIDL*GRIDL)*sizeof(float));

	//Valori delle energie
	energie = (int*)malloc(BLOCKS*sizeof(int));
	hipMalloc((void**)&energieD, BLOCKS*sizeof(int));
	for(int i=0; i<BLOCKS; i++)
		energie[i]=0;
	hipMemcpy(energieD, energie, BLOCKS*(sizeof(int)), H_D);

	//Vettore di numeri inizialmente random
	unsigned int *a, *a_d, *b, *b_d, *c, *c_d, *d, *d_d;
	float BETA = atof (pero[3]);
	
	float boltzGPU[4*DIM+1];
	for(int i=-2*DIM; i<=2*DIM; i++) boltzGPU[i+2*DIM] = min(1.0,exp(-2*BETA*J*i));
        float *boltzTex;
	hipMalloc((void**)&boltzTex, (4*DIM+1)*sizeof(float));
	hipMemcpy(boltzTex, boltzGPU,(4*DIM+1)*sizeof(float), H_D);
	hipBindTexture(NULL, boltzT, boltzTex, (4*DIM+1)*sizeof(float));

	dim3 grid(GRIDL, GRIDL/2);
	dim3 block(BLOCKL, BLOCKL/2);

	dim3 gridRES(GRIDL, GRIDL);
	dim3 blockRES(BLOCKL, BLOCKL);

	//*************Accocazione e Generazione seed iniziali                 
	a=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	b=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	c=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	d=(unsigned int*)malloc(TOT_TH*2*sizeof(unsigned int));
	
	fill_ran_vec2(a, b, c, d, TOT_TH);

	hipMalloc((void**)&a_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&b_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&c_d, 2*TOT_TH*(sizeof(unsigned int)));
	hipMalloc((void**)&d_d, 2*TOT_TH*(sizeof(unsigned int)));

	hipMemcpy(a_d, a, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(b_d, b, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(c_d, c, TOT_TH*(sizeof(unsigned int)), H_D);
	hipMemcpy(d_d, d, TOT_TH*(sizeof(unsigned int)), H_D);

	//*********Fine parte di generazione seed


	s = (spin_t*)malloc(N*sizeof(spin_t));
	for(int i=0; i<N; i++) s[i]=1;

	hipMalloc((void**)&sD, N*sizeof(spin_t));
	hipMemcpy(sD, s, N*sizeof(spin_t), H_D);
	int ie = cpu_energy(s); // Energia Iniziale.

	//printf("Energia iniziale: %i\n",ie);


	for(int i=0; i<TERM_STEP; i++){
		update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 0);
		update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 1);
		hipDeviceSynchronize();
	}


	double M=0;
	double Magnetizzazione=0;
	double E = ie;	
	int sumE=0;
	double E_2=0;
	int sub_iter=0;
	for(int i=0; i<STEP; i++){
		for(int j=0; j<VUOTO; j++){
	                update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 0);
	                update_metropolis_shared<<<grid, block>>>(sD, a_d, b_d, c_d, d_d, energieD, 1);
			hipDeviceSynchronize();
		}
		sub_iter+=VUOTO;
		Comp_Res<<<gridRES, blockRES>>>(sD, risultatiD);
		hipMemcpy(risultati, risultatiD, (GRIDL*GRIDL)*sizeof(float), D_H);

		for(int bl=0; bl < (GRIDL*GRIDL); bl++ )
			M+=risultati[bl];
		M = M / (GRIDL*GRIDL);
		
		//printf("%f\n",M);

		hipMemcpy(energie, energieD, BLOCKS*sizeof(int), D_H);
		for(int bl=0; bl < BLOCKS; bl++)
			E+=energie[bl];
		sumE += E;
		E_2 += pow(E,2.);
		E=ie;
		
		Magnetizzazione +=M;
		M=0;
	}


	E_2 = E_2/STEP;
	E = sumE/STEP ;
	Magnetizzazione/=STEP;
	
	
	
	
	
	
	
	
	
	
	if(Magnetizzazione < 0)
		Magnetizzazione*=-1;

	printf("%f\t\t%f\t\t%f\t%f\n", BETA, Magnetizzazione, E, E_2);




	hipFree(sD);
	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);
	hipFree(d_d);
	hipFree(energieD);



	return 0;



}
